#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <windows.h>

//#define BLOCK_NUM 32
//#define THREAD_NUM 256
//#define SIZE BLOCK_NUM * THREAD_NUM
//#define AREA SIZE * SIZE

#define TIME_START QueryPerformanceCounter(&start);

#define TIME_END(NAME) QueryPerformanceCounter(&end); \
printf(NAME);                                         \
printf(" takes %fs\n", (float)(end.QuadPart - start.QuadPart) / (float)freq.QuadPart);

//__global__ void matmul_improvedCUDA(const float* p1, const float* p2, float* ans) {
//    const int bid = blockIdx.x;
//    const int tid = threadIdx.x;
//
//    const int row = bid * THREAD_NUM + tid;
//    for (int i = 0; i < SIZE; i++) {
//        for (int j = 0; j < SIZE; j++) {
//            ans[row * SIZE + i] += p1[row * SIZE + j] * p2[j * SIZE + i];
//        }
//    }
//}
//
//int main() {
//    LARGE_INTEGER freq;
//    LARGE_INTEGER start;
//    LARGE_INTEGER end;
//    QueryPerformanceFrequency(&freq);
//
//
//
//    float* m1  = (float*) malloc(AREA * sizeof(float));
//    float* m2  = (float*) malloc(AREA * sizeof(float));
//    float* ans  = (float*) malloc(AREA * sizeof(float));
//
//    //initialize the test values
//    for (size_t i = 0; i < AREA; i++) {
//        m1[i] = i;
//        m2[i] = i;
//        ans[i] = 0;
//
//    }
//
//    TIME_START
//    float* gm1;
//    hipMalloc((void **) &gm1, AREA * sizeof(float));
//    float* gm2;
//    hipMalloc((void **) &gm2, AREA * sizeof(float));
//    float* gAns;
//    hipMalloc((void **) &gAns, AREA * sizeof(float));
//
//    hipMemcpy(gm1, m1, AREA * sizeof(float), hipMemcpyHostToDevice);
//    hipMemcpy(gm2, m2, AREA * sizeof(float), hipMemcpyHostToDevice);
//
//    matmul_improvedCUDA<<<BLOCK_NUM, THREAD_NUM>>>(gm1, gm2, gAns);
//
//    hipMemcpy(ans, gAns, AREA * sizeof(float), hipMemcpyDeviceToHost);
//    TIME_END("CUDA")
//}





#define BLOCK_NUM 32
#define THREAD_NUM 256
#define R_SIZE BLOCK_NUM * THREAD_NUM
#define M_SIZE R_SIZE * R_SIZE

__global__ void mat_mul(int *mat1, int *mat2, int *result) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;

    const int row = bid * THREAD_NUM + tid;
    for (int c = 0; c < R_SIZE; c++) {
        for (int n = 0; n < R_SIZE; n++) {
            result[row*R_SIZE+c] += mat1[row*R_SIZE+n] * mat2[n*R_SIZE+c];
        }
    }
}

int main(int argc, char *argv[]) {
    LARGE_INTEGER freq;
    LARGE_INTEGER start;
    LARGE_INTEGER end;
    QueryPerformanceFrequency(&freq);

    int *mat1, *mat2, *result;
    int *g_mat1, *g_mat2, *g_mat_result;

    mat1 = (int*) malloc(M_SIZE * sizeof(int));
    mat2 = (int*) malloc(M_SIZE * sizeof(int));
    result = (int*) malloc(M_SIZE * sizeof(int));

    for (int i = 0; i < M_SIZE; i++) {
        mat1[i] = rand()/1000000;
        mat2[i] = rand()/1000000;
        result[i] = 0;

    }

    TIME_START
    hipMalloc((void **)&g_mat1, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat2, sizeof(int) * M_SIZE);
    hipMalloc((void **)&g_mat_result, sizeof(int) * M_SIZE);

    hipMemcpy(g_mat1, mat1, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(g_mat2, mat2, sizeof(int) * M_SIZE, hipMemcpyHostToDevice);

    mat_mul<<<BLOCK_NUM, THREAD_NUM>>>(g_mat1, g_mat2, g_mat_result);

    hipMemcpy(result, g_mat_result, sizeof(int) * M_SIZE, hipMemcpyDeviceToHost);
    TIME_END("CUDA")
}